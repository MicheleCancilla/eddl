/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "eddl/hardware/gpu/nn/gpu_tensor_nn.h"
#include "eddl/hardware/gpu/nn/gpu_tensor_nn_kernels.h"

#include "eddl/hardware/gpu/gpu_tensor.h"

#include "eddl/tensor/tensor.h"

#define VERBOSE 0


void gpu_permute_channels_last(Tensor *A,Tensor *B)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);
  bn_permute_channels_last<<<dimGrid,dimBlock>>>(A->ptr, B->ptr,A->shape[0],A->shape[1],A->shape[2],A->shape[3],A->size);
  check_cuda(hipDeviceSynchronize(),"bn_permute_channels_last");
}

void gpu_permute_channels_first(Tensor *A,Tensor *B)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);
  bn_permute_channels_first<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,B->shape[0],B->shape[1],B->shape[2],B->shape[3],B->size);
  check_cuda(hipDeviceSynchronize(),"bn_permute_channels_first");
}

void gpu_permute_batch_last(Tensor *A,Tensor *B)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);
  bn_permute_batch_last<<<dimGrid,dimBlock>>>(A->ptr, B->ptr,A->shape[0],A->shape[1],A->shape[2],A->shape[3],A->size);
  check_cuda(hipDeviceSynchronize(),"bn_permute_batch_last");
}

void gpu_permute_batch_first(Tensor *A,Tensor *B)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);
  bn_permute_batch_first<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,B->shape[0],B->shape[1],B->shape[2],B->shape[3],B->size);
  check_cuda(hipDeviceSynchronize(),"bn_permute_batch_first");
}




















/////////
