#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.8
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "eddl/hardware/gpu/nn/gpu_tensor_nn.h"
#include "eddl/hardware/gpu/nn/gpu_tensor_nn_kernels.h"

#include "eddl/hardware/gpu/gpu_hw.h"
#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"

#ifdef cCUDNN
void * shared_workspace=nullptr;
size_t workspace_size=0;

void my_get_fdescriptor(hipdnnFilterDescriptor_t t, char * name){

    hipdnnDataType_t         dataType;
    hipdnnTensorFormat_t       format;
    int                     n;
    int                     c;
    int                     h;
    int                     w;
    check_cudnn(hipdnnGetFilter4dDescriptor(t, &dataType, &format, &n, &c, &h, &w));
    std::cout<<name<<": ("<<dataType<<", "<<n<<", "<<c<<", "<<h<<", "<<w<<")"<<std::endl;
}

void my_get_descriptor(hipdnnTensorDescriptor_t t, char * name){

    hipdnnDataType_t         dataType;
    int                     n;
    int                     c;
    int                     h;
    int                     w;
    int                     nStride;
    int                     cStride;
    int                     hStride;
    int                     wStride;
    check_cudnn(hipdnnGetTensor4dDescriptor(t, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride));
    std::cout<<name<<": ("<<dataType<<", "<<n<<", "<<c<<", "<<h<<", "<<w<<", "<<nStride<<", "<<cStride<<", "<<hStride<<", "<<wStride<<")"<<std::endl;
}



int allocate_workspace(size_t size){
    if (size <= workspace_size){
        return 0;
    }
    else {
        workspace_size = size;
        hipFree(shared_workspace);
        return hipMalloc((void **) &shared_workspace, size);
    }
}
#endif

void gpu_im2col(ConvolDescriptor *D, int col2im){
  int device=D->I->gpu_device;
  hipSetDevice(device);

  setDims(D->gpuI)
  dimGrid.x*=D->I->shape[0];

  if (col2im)
    gpu_im2col_k<<<dimGrid,dimBlock>>>(D->ID->ptr, D->gpuI->ptr,D->I->shape[0],D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,1);
  else
    gpu_im2col_k<<<dimGrid,dimBlock>>>(D->I->ptr, D->gpuI->ptr,D->I->shape[0],D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,0);

  check_cuda(hipDeviceSynchronize(),"gpu_im2col");

}

void gpu_im2col_low(ConvolDescriptor *D, int col2im,int b){
  int device=D->I->gpu_device;
  hipSetDevice(device);

  setDims(D->gpuI)

  if (col2im)
    gpu_im2col_k_low<<<dimGrid,dimBlock>>>(D->ID->ptr, b, D->gpuI->ptr,D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,1);
  else
    gpu_im2col_k_low<<<dimGrid,dimBlock>>>(D->I->ptr, b, D->gpuI->ptr,D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,0);

  check_cuda(hipDeviceSynchronize(),"gpu_im2col");

}




void gpu_conv2D(ConvolDescriptor *D) {

  int device=D->I->gpu_device;
  hipSetDevice(device);
  float alpha = 1.0f;
  float beta = 0.0f;

#ifndef cCUDNN
  int osize=D->z*D->r*D->c;
  int isize=D->kz*D->kr*D->kc*D->r*D->c;
  D->gpuK->ptr=D->K->ptr;
  D->gpuO->ptr=D->O->ptr;
  D->gpuI->ptr=D->gpuIB->ptr;


  if (D->mem_level>1) {
    for(int b=0;b<D->I->shape[0];b++,D->gpuO->ptr+=osize) {
      gpu_im2col_low(D,0,b);
      gpu_mult2D(D->gpuK,0,D->gpuI,1,D->gpuO,0);
    }
  }
  else {

    gpu_im2col(D,0);
    if (D->mem_level==0) {
      gpu_mult2D(D->gpuK,0,D->gpuIB,1,D->gpuOB,0);
      setDims(D->O);
      gpu_traspose_batch_depth<<<dimGrid,dimBlock>>>(D->gpuOB->ptr, D->O->ptr, D->O->shape[0], D->z, D->r, D->c);
      check_cuda(hipDeviceSynchronize(),"gpu_batch_depth");
    }
    else {
      gpu_im2col(D,0);
      for(int b=0;b<D->I->shape[0];b++,D->gpuO->ptr+=osize,D->gpuI->ptr+=isize)
        gpu_mult2D(D->gpuK,0,D->gpuI,1,D->gpuO,0);
    }

  }
#else
  // FWD environment
  if (D->cudnn_env_init < 0){
      D->cudnn_env_init = 1;

      int requestedAlgoCount;
      check_cudnn(cudnnGetConvolutionForwardAlgorithmMaxCount( D->cudnn_handle, &requestedAlgoCount));

      int returnedAlgoCount;
      hipdnnConvolutionFwdAlgoPerf_t * perfResults = new hipdnnConvolutionFwdAlgoPerf_t [requestedAlgoCount];

      check_cudnn(hipdnnFindConvolutionForwardAlgorithm( D->cudnn_handle, D->xDesc, D->wDesc, D->convolution_descriptor, D->yDesc,
                  requestedAlgoCount, &returnedAlgoCount, perfResults));

      int aux_alg = 0;
      size_t size;
      do{
          D->fwd_algorithm = perfResults[aux_alg].algo;

          check_cudnn(hipdnnGetConvolutionForwardWorkspaceSize(D->cudnn_handle,D->xDesc, D->wDesc,
                                                              D->convolution_descriptor,  D->yDesc,
                                                              D->fwd_algorithm, &size));
          aux_alg++;
      }
      while(allocate_workspace(size));
  }
  //BWD environment
  if (D->cudnn_conv_back_init < 0){
      D->cudnn_conv_back_init = 1;
       int requestedAlgoCount;

      check_cudnn(cudnnGetConvolutionBackwardFilterAlgorithmMaxCount(
              D->cudnn_handle, &requestedAlgoCount));
      int returnedAlgoCount;
      hipdnnConvolutionBwdFilterAlgoPerf_t * perfResults = new hipdnnConvolutionBwdFilterAlgoPerf_t [requestedAlgoCount];

      check_cudnn(hipdnnFindConvolutionBackwardFilterAlgorithm(D->cudnn_handle, D->xDesc, D->yDesc,
                                                        D->convolution_descriptor, D->wDesc, requestedAlgoCount,
                                                        &returnedAlgoCount, perfResults));
      int aux_alg = 0;
      size_t size;
      do{
          D->bwd_filter_algorithm = perfResults[aux_alg].algo;

          check_cudnn(hipdnnGetConvolutionBackwardFilterWorkspaceSize(D->cudnn_handle,D->xDesc, D->yDesc,
                                                              D->convolution_descriptor,  D->wDesc,
                                                              D->bwd_filter_algorithm, &size));
          aux_alg++;
      }
      while(allocate_workspace(size));

      //////////// DATA!!!!
      requestedAlgoCount = 0;
     check_cudnn(cudnnGetConvolutionBackwardDataAlgorithmMaxCount(D->cudnn_handle, &requestedAlgoCount));
     returnedAlgoCount=0;
      hipdnnConvolutionBwdDataAlgoPerf_t * perfResults_d = new hipdnnConvolutionBwdDataAlgoPerf_t [requestedAlgoCount];

      check_cudnn(hipdnnFindConvolutionBackwardDataAlgorithm(D->cudnn_handle, D->wDesc, D->yDesc,
                                                        D->convolution_descriptor, D->xDesc, requestedAlgoCount,
                                                        &returnedAlgoCount, perfResults_d));
      aux_alg = 0;
       size=0;
      do{
          D->bwd_data_algorithm = perfResults_d[aux_alg].algo;

          check_cudnn(hipdnnGetConvolutionBackwardDataWorkspaceSize(D->cudnn_handle,D->wDesc, D->yDesc,
                                                              D->convolution_descriptor,  D->xDesc,
                                                              D->bwd_data_algorithm, &size));
          aux_alg++;
      }
      while(allocate_workspace(size));

  }
  check_cudnn(hipdnnConvolutionForward( D->cudnn_handle, &alpha, D->xDesc, D->I->ptr,
                                       D->wDesc, D->K->ptr,
                                       D->convolution_descriptor, D->fwd_algorithm,
                                       shared_workspace, workspace_size,
                                       &beta, D->yDesc, D->O->ptr));
#endif
  if (D->use_bias) {
#ifndef cCUDNN
    int size=D->bias->shape[0];
    for(int i=0;i<size;i+=1024) {
      int s=min(1024,size-i);
      gpu_addbias_k<<<D->O->shape[0],s>>>(D->O->ptr, D->O->shape[0], D->r,D->c,D->nk,D->bias->ptr,i);
      check_cuda(hipDeviceSynchronize(),"gpu_addbias");
    }
#else
    check_cudnn(hipdnnAddTensor(D->cudnn_handle, &alpha, D->bDesc, D->bias->ptr,
                               &alpha, D->yDesc, D->O->ptr));
#endif
  }


}


void gpu_conv2D_grad(ConvolDescriptor *D){

  int device=D->I->gpu_device;

  hipSetDevice(device);
  float alpha=1.0;
  float beta = 0.0;
#ifndef cCUDNN
  int osize=D->z*D->r*D->c;
  int isize=D->kz*D->kr*D->kc*D->r*D->c;

  D->gpugK->ptr=D->gK->ptr;
  D->gpuD->ptr=D->D->ptr;
  D->gpuI->ptr=D->gpuIB->ptr;

  if (D->mem_level>1) {
    for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize){
      gpu_im2col_low(D,0,b);
      gpu_mult2D(D->gpuD,0,D->gpuI,0,D->gpugK,1);
    }
  }
  else {
    if (D->mem_level==0) {
      setDims(D->D);
      gpu_traspose_batch_depth<<<dimGrid,dimBlock>>>(D->D->ptr, D->gpuOB->ptr, D->z, D->O->shape[0], D->r, D->c);
      check_cuda(hipDeviceSynchronize(),"gpu_batch_depth");

      gpu_mult2D(D->gpuOB,0,D->gpuIB,0,D->gpugK,1);
    }
    else {
      for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize,D->gpuI->ptr+=isize)
        gpu_mult2D(D->gpuD,0,D->gpuI,0,D->gpugK,1);
    }
  }
#else
        check_cudnn(hipdnnConvolutionBackwardFilter(D->cudnn_handle, &alpha,
                                      D->xDesc, D->I->ptr,
                                      D->yDesc, D->D->ptr, D->convolution_descriptor,
                                      D->bwd_filter_algorithm,
                                      shared_workspace, workspace_size,
                                      &beta, D->wDesc, D->gK->ptr));

#endif
  if (D->use_bias) {
#ifndef cCUDNN
    int size=D->bias->shape[0];
    for(int i=0;i<size;i+=1024) {
      int s=min(1024,size-i);
      gpu_deltabias_k<<<D->D->shape[0],s>>>(D->D->ptr, D->D->shape[0], D->r,D->c,D->nk,D->gbias->ptr,i);
      check_cuda(hipDeviceSynchronize(),"gpu_deltabias");
    }
#else
      check_cudnn(hipdnnConvolutionBackwardBias(D->cudnn_handle, &alpha, D->yDesc, D->D->ptr,
                                               &beta, D->bDesc, D->gbias->ptr));
#endif

  }


}


void gpu_conv2D_back(ConvolDescriptor *D){


  int device=D->I->gpu_device;
  hipSetDevice(device);
#ifndef cCUDNN
  int osize=D->z*D->r*D->c;
  int isize=D->kz*D->kr*D->kc*D->r*D->c;
  D->gpuK->ptr=D->K->ptr;
  D->gpuD->ptr=D->D->ptr;
  D->gpuI->ptr=D->gpuIB->ptr;


  if (D->mem_level>1) {
    for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize) {
        gpu_mult2D(D->gpuD, 1, D->gpuK, 0, D->gpuI, 0);
        gpu_im2col_low(D,1,b);
    }
  }
  else {
    if (D->mem_level==0) {
      setDims(D->D);
      gpu_traspose_batch_depth<<<dimGrid,dimBlock>>>(D->D->ptr, D->gpuOB->ptr,  D->z, D->O->shape[0],D->r, D->c);
      check_cuda(hipDeviceSynchronize(),"gpu_batch_depth");

      gpu_mult2D(D->gpuOB, 1, D->gpuK, 0, D->gpuIB, 0);
      D->gpuI->ptr=D->gpuIB->ptr;
      gpu_im2col(D,1);
    }
    else{
      for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize,D->gpuI->ptr+=isize) {
          gpu_mult2D(D->gpuD, 1, D->gpuK, 0, D->gpuI, 0);
      }
      D->gpuI->ptr=D->gpuIB->ptr;
      gpu_im2col(D,1);
    }
  }
#else
    float alpha = 1.0f;
    float beta = 1.0f;
    check_cudnn(hipdnnConvolutionBackwardData(D->cudnn_handle, &alpha, D->wDesc, D->K->ptr,
                                             D->yDesc, D->D->ptr,
                                             D->convolution_descriptor, D->bwd_data_algorithm,
                                             shared_workspace, workspace_size,
                                             &beta, D->xDesc, D->ID->ptr));
#endif

}


void gpu_conv3D(ConvolDescriptor3D *D){

}

void gpu_conv3D_grad(ConvolDescriptor3D *D){

}

void gpu_conv3D_back(ConvolDescriptor3D *D){

}
