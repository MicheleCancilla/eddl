#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#include "eddl/hardware/gpu/nn/gpu_tensor_nn_kernels.h"
#include "eddl/hardware/gpu/gpu_kernels.h"


__global__ void  gpu_traspose_batch_depth(float *ptrB, float *ptr, int b,int z,int r, int c)
{
  long int ops=b*z*r*c;
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;


  if (thread_id_x < ops) {
    int bo=thread_id_x/(z*r*c);
    int zom=thread_id_x%(z*r*c);
    int zo=zom/(r*c);
    int rom=zom%(r*c);
    int ro=rom/c;
    int co=rom%c;

    int pos=(zo*(b*r*c))+(bo*(r*c))+(ro*c)+co;

    ptr[thread_id_x]=ptrB[pos];

  }

}

__global__ void  gpu_addbias_k(float *O, int batch, int r,int c,int nk,float *bias,int offset)
{
  int size=nk*r*c;
  int thread_id_x=threadIdx.x;

  int p=blockIdx.x*size+(thread_id_x+offset)*r*c;
  for (int i = 0; i < r*c; i++)
     O[p+i]+=bias[thread_id_x+offset];

}

__global__ void  gpu_deltabias_k(float *D, int batch, int r,int c,int nk,float *bias, int offset)
{
  int size=nk*r*c;
  int thread_id_x=threadIdx.x;

  int p=blockIdx.x*size+(thread_id_x+offset)*r*c;
  for (int i = 0; i < r*c; i++)
    atomicAdd(&(bias[thread_id_x+offset]),D[p+i]);

}


__global__ void gpu_im2col_k(float* I, float *ptrI,int batch,int irows,int icols, int idepth, float* K, int nk, int kr,int kc, float* O,int orows,int ocols,int sr,int sc,int padrt,int padrb,int padcl,int padcr,int col2im)
{
  long int ops=batch*orows*ocols*kr*kc*idepth;
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;


  if (thread_id_x < ops) {
    int iz,ix,iy;

    int ksize=kr*kc*idepth;

    int im=thread_id_x/(ksize*orows*ocols);
    int ioffset=im*irows*icols*idepth;


    int tx=thread_id_x%(ksize*orows*ocols);


    int r=tx/ksize;
    int c=tx%ksize;

    int oy=r/ocols;
    int ox=r%ocols;

    ix=(ox*sc)-padcl;
    iy=(oy*sr)-padrt;
    iz=c/(kr*kc);

    c=c%(kr*kc);

    iy+=c/kc;
    ix+=c%kc;

    if ((ix>=0)&&(ix<icols)&&(iy>=0)&&(iy<irows)) {
      int p=iz*(irows*icols)+(iy*icols)+ix;
      if (col2im)
        atomicAdd(&(I[p+ioffset]),ptrI[thread_id_x]);
      else
	ptrI[thread_id_x]=I[p+ioffset];
    }
    else
      if (!col2im)
        ptrI[thread_id_x]=0;

  }

}

__global__ void gpu_im2col_k_low(float* I, int b, float *ptrI,int irows,int icols, int idepth, float* K, int nk, int kr,int kc, float* O,int orows,int ocols,int sr,int sc,int padrt,int padrb,int padcl,int padcr,int col2im)
{
  long int ops=orows*ocols*kr*kc*idepth;
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;


  if (thread_id_x < ops) {
    int iz,ix,iy;

    int ksize=kr*kc*idepth;

    int im=b;
    int ioffset=im*irows*icols*idepth;


    int tx=thread_id_x%(ksize*orows*ocols);


    int r=tx/ksize;
    int c=tx%ksize;

    int oy=r/ocols;
    int ox=r%ocols;

    ix=(ox*sc)-padcl;
    iy=(oy*sr)-padrt;
    iz=c/(kr*kc);

    c=c%(kr*kc);

    iy+=c/kc;
    ix+=c%kc;

    if ((ix>=0)&&(ix<icols)&&(iy>=0)&&(iy<irows)) {
      int p=iz*(irows*icols)+(iy*icols)+ix;
      if (col2im)
        atomicAdd(&(I[p+ioffset]),ptrI[thread_id_x]);
      else
      	ptrI[thread_id_x]=I[p+ioffset];
    }
    else
      if (!col2im)
        ptrI[thread_id_x]=0;

  }

}

__global__ void gpu_low_mem_conv3D(int batch_size,
        int channels, int image_depth, int image_rows, int image_cols, const float *image,
        int num_kernels, int kernel_depth, int kernel_rows, int kernel_cols, const float *kernel,
        int out_depth, int out_rows, int out_cols, float *output,
        int pad_depth, int pad_row, int pad_col,
        int stride_depth, int stride_rows, int stride_cols)
{
    // for (int b = 0; b < batch_size; b++)
    int b = blockIdx.y;
    /* for (int nk = 0; nk < num_kernels; nk++)
    for (int k = 0; k < out_depth; k++)
    for (int i = 0; i < out_rows; i++)
    for (int j = 0; j < out_cols; j++) { */
    int nk = blockIdx.x * low_mem_block_size + threadIdx.x;
    int output_size = num_kernels * out_depth * out_rows * out_cols;
    if (nk >= output_size) return;
    int j = nk % out_cols; nk /= out_cols;
    int i = nk % out_rows; nk /= out_rows;
    int k = nk % out_depth; nk /= out_depth;

    float s = 0;
    for (int z = 0; z < kernel_depth; z++) {
        int pz = k * stride_depth + z - pad_depth;
        if (pz >= 0 && pz < image_depth)
        for (int x = 0; x < kernel_rows; x++) {
            int px = i * stride_rows + x - pad_row;
            if (px >= 0 && px < image_rows)
            for (int y = 0; y < kernel_cols; y++) {
                int py = j * stride_cols + y - pad_col;
                if (py >= 0 && py < image_cols) {
                    for (int c = 0; c < channels; c++)
                        s += kernel[(((nk * channels + c) * kernel_depth + z) * kernel_rows + x) * kernel_cols + y]
                           * image[(((b * channels + c) * image_depth + pz) * image_rows + px) * image_cols + py];
                }
            }
        }
    }
    output[(((b * num_kernels + nk) * out_depth + k) * out_rows + i) * out_cols + j] = s;
}

__global__ void gpu_low_mem_conv2D_grad(int batch_size,
        int channels, int image_rows, int image_cols, const float *image,
        int num_kernels, int kernel_rows, int kernel_cols, float *kernel,
        int out_rows, int out_cols, const float *delta,
        int pad_row, int pad_col,
        int stride_rows, int stride_cols)
{
    // for (int b = 0; b < batch_size; b++) {
    int b = blockIdx.y;
    /* for (int nk = 0; nk < num_kernels; nk++)
    for (int c = 0; c < channels; c++)
    for (int x = 0; x < kernel_rows; x++)
    for (int y = 0; y < kernel_cols; y++) { */
    int nk = blockIdx.x * low_mem_block_size + threadIdx.x;
    int kernel_size = num_kernels * channels * kernel_rows * kernel_cols;
    if (nk >= kernel_size) return;
    int y = nk % kernel_cols; nk /= kernel_cols;
    int x = nk % kernel_rows; nk /= kernel_rows;
    int c = nk % channels; nk /= channels;

    float s = 0.0;
    for (int i = 0; i < out_rows; i++) {
        int px = i * stride_rows - pad_row + x;
        if (px < 0) continue;
        if (px >= image_rows) continue;
        for (int j = 0; j < out_cols; j++) {
            int py = j * stride_cols - pad_col + y;
            if (py < 0) continue;
            if (py >= image_cols) continue;
            s += image[((b * channels + c) * image_rows + px) * image_cols + py] *
                delta[((b * num_kernels + nk) * out_rows + i) * out_cols + j];
        }
    }
    // kernel[(((nk * channels + c) * kernel_rows + x) * kernel_cols) + y] = s;
    atomicAdd(kernel + blockIdx.x * low_mem_block_size + threadIdx.x, s);
}

__global__ void gpu_low_mem_conv2D_back(int batch_size,
        int channels, int image_rows, int image_cols, float *image,
        int num_kernels, int kernel_rows, int kernel_cols, const float *kernel,
        int out_rows, int out_cols, const float *delta,
        int pad_row, int pad_col,
        int stride_rows, int stride_cols)
{
    // for (int b = 0; b < batch_size; b++)
    int b = blockIdx.y;
    /* for (int c = 0; c < channels; c++)
    for (int i = 0; i < out_rows; i++)
    for (int j = 0; j < out_cols; j++) */
    int c = blockIdx.x * low_mem_block_size + threadIdx.x;
    int image_size = channels * out_rows * out_cols;
    if (c >= image_size) return;
    int j = c % out_cols; c /= out_cols;
    int i = c % out_rows; c /= out_rows;

    for (int x = 0; x < kernel_rows; x++) {
        int px = i * stride_rows - pad_row + x;
        if (px < 0) continue;
        if (px >= image_rows) continue;
        for (int y = 0; y < kernel_cols; y++) {
            int py = j * stride_cols - pad_col + y;
            if (py < 0) continue;
            if (py >= image_cols) continue;
            float s = 0.0;
            for (int nk = 0; nk < num_kernels; nk++)
                s += delta[((b * num_kernels + nk) * out_rows + i) * out_cols + j]
                   * kernel[((nk * channels + c) * kernel_rows + x) * kernel_cols + y];
            atomicAdd(image + ((b * channels + c) * image_rows + px) * image_cols + py, s);
        }
    }
}
